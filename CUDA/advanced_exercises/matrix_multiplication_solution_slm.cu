
#include <hip/hip_runtime.h>
    #define THREAD_INDEX (threadIdx.y * blockDim.x + threadIdx.x)

    __global__ void matmul(float *A, float *B, float *C, int N) {
        float sum = 0;
        int2 global_id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                   blockIdx.y * blockDim.y + threadIdx.y);
                                   
        if (global_id.x >= N || global_id.y >= N) {
            return;
        }
                
        extern __shared__ float slm[];
        float *slm_A = &slm[0];
        float *slm_B = &slm[blockDim.x * blockDim.y];
        
        for (int b = 0; b < gridDim.x; b++) {
            int2 gidA = make_int2(b * blockDim.x + threadIdx.x,  global_id.y);
            int2 gidB = make_int2(global_id.x,  b * blockDim.y + threadIdx.y);
                        
            slm_A[THREAD_INDEX] = A[gidA.y * N + gidA.x];
            slm_B[THREAD_INDEX] = B[gidB.y * N + gidB.x];
                
            __syncthreads();
            
            for (int i = 0; i < blockDim.x; i++) {
                sum += slm_A[blockDim.x * threadIdx.y + i] * slm_B[blockDim.x * i + threadIdx.x];
            }            
            
            __syncthreads();
        }
        
        C[global_id.y * N + global_id.x] = sum;
    }
